#include "hip/hip_runtime.h"
#include <map>
#include <deque>
#include <vector>
#include <sstream>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sys/time.h>

#ifndef __HIPCC__
#define XCPU
#endif

#ifdef XCPU
#include <cmath>
#include <cstring>
#endif

#ifdef USE_I3_LOGGING
#include "icetray/I3Logging.h"
#else
#define log_info_stream(msg) \
  do { std::cerr << msg << std::endl; } while (0)
#endif

using namespace std;

namespace xppc{
  #include "ini.cxx"
  #include "pro.cu"

  void initialize(float enh = 1.f){ m.set(); q.eff*=enh; }

  unsigned int pmax, pmxo, pn, pk, hquo;

  void setq(){
    char * HQUO=getenv("HQUO");
    hquo=HQUO==NULL?1:atoi(HQUO);
    cerr<<"HQUO(photons/max number of hits)="<<hquo<<endl;
  }

  dats *e;  // pointer to a copy of "d" on device
  int nblk, nthr, ntot;

  void ini(){
    setq();
    rs_ini();
    pn=0; pk=0;

    ntot=nblk*nthr;
    pmax=ntot*NPHO;
    pmxo=pmax/OVER;
    pmax=pmxo*OVER;
    d.hnum=pmax/hquo;

    d.gdev=0; d.gnum=1;
    d.gini=0; d.gspc=pmax; d.gtot=pmax; d.gdiv=1;

    {
      d.hits = q.hits = new hit[d.hnum];
      d.pz = q.pz = new photon[pmxo];
      d.bf = new pbuf[pmax];
    }

    {
      d.z=&z; d.oms=q.oms; e=&d;
    }

    {
      unsigned int size=d.rsize, need=seed+1;
      if(size<need) cerr<<"Error: not enough multipliers: asked for "<<seed<<"-th out of "<<size<<"!"<<endl;
    }
  }

  void fin(){
    delete d.pz;
    delete d.hits;
    delete d.bf;
  }




  void print();

  void kernel(unsigned int num){
    unsigned int & old = num;
    if(old>0){
      d.hidx=0;
      for(d.blockIdx=0, d.gridDim=nblk, blockDim.x=nthr; d.blockIdx<d.gridDim; d.blockIdx++)
	    for(threadIdx.x=0; threadIdx.x<blockDim.x; threadIdx.x++) propagate(e, num);  
      if(d.hidx>d.hnum){ cerr<<"Error: data buffer overflow occurred: "<<d.hidx<<">"<<d.hnum<<"!"<<endl; d.hidx=d.hnum; }
      log_info_stream("photons: "<<old<<"  hits: "<<d.hidx);
    }



  if(old>0) print();

  }


  void start(){}
  void stop(){}
  void choose(int device){
    sv+=device;
    seed=device;
    nblk=NBLK, nthr=NTHR;
  }
  void listDevices(){}


  #include "f2k.cxx"
}


using namespace xppc;

float zdh;

float zshift(float4 r){
  zdh=d.dh;
  return zshift(d, r, zdh);
}

int main(int arg_c, char *arg_a[]){
  start();
  if(arg_c<=1){
    listDevices();
    fprintf(stderr, "Use: %s [device] (f2k muons)\n"
	    "     %s [str] [om] [num] [device] (flasher)\n", arg_a[0], arg_a[0]);
  }
  else if(0==strcmp(arg_a[1], "-")){
    initialize();
    ices & w = z.w[WNUM/2];
    cerr<<"For wavelength="<<q.wvs[w.wvl].w<<" [nm]  np="<<(1/w.coschr)<<"  cm="<<1/w.ocm<<" [m/ns]"<<endl;
    float4 r;
    r.w=0;
    if(arg_c==4){
      r.x=atof(arg_a[2]);
      r.y=atof(arg_a[3]);
    }
    else r.x=0, r.y=0;
    for(int i=0; i<d.size; i++){
      float z=d.hmin+d.dh*i;
      r.z=z; for(int j=0; j<10; j++) r.z=z+zshift(r); z=r.z;
      cout<<z<<" "<<w.z[i].abs<<" "<<w.z[i].sca*(1-d.g)<<" "<<d.az[i].ra*d.sum<<endl;
    }
  }
  else if(0==strcmp(arg_a[1], "=")){
    initialize();
    ices & w = z.w[WNUM/2];
    cerr<<"For wavelength="<<q.wvs[w.wvl].w<<" [nm]  np="<<(1/w.coschr)<<"  cm="<<1/w.ocm<<" [m/ns]"<<endl;
    float4 r;
    r.w=0;
    string in;
    while(getline(cin, in)){
      if(3==sscanf(in.c_str(), "%f %f %f", &r.x, &r.y, &r.z)){
	float dz=zshift(r);
	cout<<in<<" "<<dz<<" "<<zdh<<endl;
      }
    }
  }
  else if(0==strcmp(arg_a[1], "_")){
    initialize();
    float4 r;
    r.w=0;
    for(r.x=-750.f; r.x<751.f; r.x+=3.f) for(r.y=-750.f; r.y<751.f; r.y+=3.f) for(float z=-750.f; z<751.f; z+=6.f){
	  r.z=z; for(int j=0; j<10; j++) r.z=z+zshift(r);
	  cout<<z<<" "<<r.x<<" "<<r.y<<" "<<(r.z-z)<<endl;
	}
  }
  else if(arg_c<=2){
    int device=0;
    if(arg_c>1) device=atoi(arg_a[1]);
    initialize();
    choose(device);
    fprintf(stderr, "Processing f2k muons from stdin on device %d\n", device);
    f2k();
  }
  else{
    int str=0, dom=0, device=0, itr=0;
    unsigned long long num=1000000ULL;

    if(arg_c>1) str=atoi(arg_a[1]);
    if(arg_c>2) dom=atoi(arg_a[2]);
    if(arg_c>3){
      num=(unsigned long long) atof(arg_a[3]);
      char * sub = strchr(arg_a[3], '*');
      if(sub!=NULL) itr=(int) atof(++sub);
    }
    if(arg_c>4) device=atoi(arg_a[4]);
    initialize();
    choose(device);
    fprintf(stderr, "Running flasher simulation on device %d\n", device);
    flasher(str, dom, num, itr);
  }

  stop();
}