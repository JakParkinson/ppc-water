#include "hip/hip_runtime.h"
#include <map>
#include <deque>
#include <vector>
#include <sstream>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sys/time.h>

#ifndef __HIPCC__
#define XCPU
#endif

#ifdef XCPU
#include <cmath>
#include <cstring>
#endif



using namespace std;

namespace xppc{

#include "ini.cxx"
#include "pro.cu"  // You might need to adapt this later

  void initialize(float enh = 1.f){ m.set(); q.eff*=enh; }

  unsigned int pmax, pmxo, pn, pk, hquo;

  void setq(){
    char * HQUO=getenv("HQUO");
    hquo=HQUO==NULL?1:atoi(HQUO);
    cerr<<"HQUO(photons/max number of hits)="<<hquo<<endl;
  }

  // Keep only this XCPU section, remove everything in the #else
  dats *e;
  int nblk, nthr, ntot;

  void ini(){
    setq();
    rs_ini();
    pn=0; pk=0;

    ntot=nblk*nthr;
    pmax=ntot*NPHO;
    pmxo=pmax/OVER;
    pmax=pmxo*OVER;
    d.hnum=pmax/hquo;

    d.gdev=0; d.gnum=1;
    d.gini=0; d.gspc=pmax; d.gtot=pmax; d.gdiv=1;

    {
      d.hits = q.hits = new hit[d.hnum];
      d.pz = q.pz = new photon[pmxo];
      d.bf = new pbuf[pmax];
    }

    {
      d.z=&z; d.oms=q.oms; e=&d;
    }

    {
      unsigned int size=d.rsize, need=seed+1;
      if(size<need) cerr<<"Error: not enough multipliers: asked for "<<seed<<"-th out of "<<size<<"!"<<endl;
    }
  }

  void fin(){
    delete d.pz;
    delete d.hits;
    delete d.bf;
  }


  void start(){}
  void stop(){}
  void choose(int device){
    sv+=device;
    seed=device;
    nblk=NBLK, nthr=NTHR;
  }
  void listDevices(){}

#include "f2k.cxx"
}

#ifndef XLIB
using namespace xppc;

float zdh;

float zshift(float4 r){
  zdh=d.dh;
  return zshift(d, r, zdh);
}

// void initialize(){
//     m.set();
// }

float photon_yield(string loss_type, int energy_gev, float track_length) {
    float rho = 0.9216f; // density of ice in icecube
    float logE = logf(energy_gev);
    float em_cascade_value=5.321*0.910f/rho;  // important for em cascade photons
    float eff_tracl_length = 0;
    float num_photons = 0.0f;
    float ppm = 2000.0f;
    if (loss_type == "amu-") {
        // calcualte photons for muon track
        float additional_track = 1+ max(0.0f, 0.1880f+0.0206f*logE)*0.910f/rho;
        num_photons = track_length>0?track_length*additional_track:0;
    } 
    if (loss_type == "em") {
        num_photons=energy_gev*em_cascade_value;
    }
    return num_photons*ppm;
}

int main(int argc, char* argv[]) {
  start(); // doesnt do anythin i think
    // Arguements: loss type str, energy gev, track length m
    cout << "initalizing: " << endl;
    initialize();
    string loss_type= argv[1];
    int energy_gev = stoi(argv[2]);
    float track_length = stof(argv[3]);

    cout << "loss type: " << loss_type << ", has energy: " << energy_gev << ", and track length: " << track_length << endl;
    float photons = photon_yield(loss_type, energy_gev, track_length);
    cout << "photons: " << photons << endl;

    return 0;
}
#endif

